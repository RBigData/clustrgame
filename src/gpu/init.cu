#include <hip/hip_runtime.h>
#include <mpi.h>

extern "C" {
#include <Rinternals.h>
#include "../mpi_utils.h"
}


extern "C" SEXP R_clustrgame_init(SEXP comm_)
{
  int ngpus;
  int rank;
  int id;
  
  MPI_Comm comm = get_mpi_comm_from_Robj(comm_);
  MPI_Comm_rank(comm, &rank);
  
  hipGetDeviceCount(&ngpus);
  
  id = rank % ngpus;
  hipSetDevice(id);
  
  return R_NilValue;
}
